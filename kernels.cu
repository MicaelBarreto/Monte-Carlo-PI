#include "hip/hip_runtime.h"
#include "kernels.cuh"




__global__ void setup_kernel(hiprandState *worker_state)
{
	int index = threadIdx.x + blockDim.x*blockIdx.x;
    hiprand_init(123456789, index, 0, &worker_state[index]);
}




__global__ void monti_carlo_pi_kernel(hiprandState *worker_state, int *count, int m)
{
	unsigned int index = threadIdx.x + blockDim.x*blockIdx.x;

	__shared__ int cache[256];
	cache[threadIdx.x] = 0;
	__syncthreads();


	unsigned int temp = 0;
	while(temp < m){
		float x = hiprand_uniform(&worker_state[index]);
		float y = hiprand_uniform(&worker_state[index]);
		float r = x*x + y*y;

		if(r <= 1){
			cache[threadIdx.x]++;
		}
		temp++; 
	}


	// Reduction
	int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] += cache[threadIdx.x + i];
		}

		i /= 2;
		__syncthreads();
	}


	// Update count
	if(threadIdx.x == 0){
		atomicAdd(count, cache[0]);
	}
}