#include <iostream>
#include "kernels.cuh"
#include <hiprand.h>
#include <time.h>
#include <random>
#include <math.h>




int main()
{
	int r = 128; // Radius
	int n = 20; // Threads Number
	int m = 250000; // Times 4 each Thread
	int *master_count;
	int *worker_count;
	hiprandState *worker_state;
	float pi;


	// Allocate variables on memory
	master_count = (int*)malloc(n*sizeof(int));
	hipMalloc((void**)&worker_count, n*sizeof(int));
	hipMalloc((void**)&worker_state, n*sizeof(hiprandState));
	hipMemset(worker_count, 0, sizeof(int));


	// Timing
	float elapsed_time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	// Set Worker Kernel
	dim3 gridSize = r;
	dim3 blockSize = r;
	setup_kernel<<<gridSize, blockSize>>>(worker_state);


	// Start Worker Monte Carlo Kernel
	monti_carlo_pi_kernel<<<gridSize, blockSize>>>(worker_state, worker_count, m);


	// Get Results from Workers
	hipMemcpy(master_count, worker_count, sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	// Calculate && Prompt Results
	pi = *master_count*4.0/(n*m);
	std::cout<<"Monte Carlo PI Approximation calculated on GPU was "<<pi<<" and took "<<elapsed_time<<" ms to calculate "<<std::endl;


	// Free Memory
	free(master_count);
	hipFree(worker_count);
	hipFree(worker_state);
}
